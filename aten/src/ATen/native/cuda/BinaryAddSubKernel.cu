#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/BinaryOps.h>

// TODO: update to use lazynvrtc
#include <ATen/cuda/nvrtc_stub/ATenNVRTC.h>
#include <torch/csrc/jit/resource_guard.h>
#include <sstream>
#include <torch/csrc/jit/frontend/code_template.h>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

template<typename scalar_t>
struct AddFunctor {
  AddFunctor(scalar_t a): alpha(a) {}
  __device__ __forceinline__ scalar_t operator() (const scalar_t a, const scalar_t b) const {
    return a + alpha * b;
  }
  private:
    scalar_t alpha;
};

void add_kernel_cuda(TensorIterator& iter, Scalar alpha_scalar) {
  // create template here
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(kHalf, kBool, kBFloat16, iter.common_dtype(), "add_cuda/sub_cuda", [&]() {
    // NOTE: we don't need compile-time switching this does at all, so maybe use alternative?
    // Question: is instantiating worthwhile vs. just recompiling?
      // Cons of recompilation: string manipulation done every time
      // Cons of recompilation: need your own code template
      // Cons of instantiation: complicated
    // instantiate dispatched scalar types using the template here
    // this happens at runtime before the call
    // cache whether instantiated or not
    // call
    AddFunctor<scalar_t> f(alpha_scalar.to<scalar_t>());
    gpu_kernel_with_scalars(iter, f);
  });
}

static void sub_kernel_cuda(TensorIterator& iter, Scalar alpha_scalar) {
  add_kernel_cuda(iter, -alpha_scalar);
}

REGISTER_DISPATCH(add_stub, &add_kernel_cuda);
REGISTER_DISPATCH(sub_stub, &sub_kernel_cuda);

// TODO: update this
static void getMajorMinor(
    const hipDeviceProp_t* const prop,
    int& major,
    int& minor) {
  int nvrtc_major, nvrtc_minor;
  AT_CUDA_NVRTC_CHECK(at::globalContext().getNVRTC().hiprtcVersion(&nvrtc_major, &nvrtc_minor));

  // Short-circuits if NVRTC version too low
  AT_ASSERT(nvrtc_major >= 6);

  // Major and minor is determined by device properties and
  // possibly "downcompiled" to a lower (compatible) compute architecture
  // based on the NVRTC version
  major = prop->major;
  minor = prop->minor;
  if (nvrtc_major <= 7 && prop->major > 5) { // 7 supports 2-5.x
    major = 5;
    minor = 0;
  } else if (nvrtc_major <= 8 && prop->major > 6) { // 8 supports 2-6.x
    major = 6;
    minor = 0;
  } else if (nvrtc_major <= 9 && prop->major >= 7) { // 9 supports 3-7.2
    major = 7;
    if (prop->major == 7 && prop->minor <= 2)
      minor = prop->minor;
    else
      minor = 0;
  } else if (nvrtc_major <= 10 && prop->major >= 7) { // 10 supports 3-7.5
    major = 7;
    if (prop->major == 7 && prop->minor <= 5) {
      minor = prop->minor;
    } else {
      minor = 0;
    }
  }
}

namespace {

  // TODO: update with IntDivider
  // TODO: possibly size specialize with template
struct TensorAccessor {
  TensorAccessor(
      const IntArrayRef shape,
      const IntArrayRef strides,
      const int64_t _element_size)
      : element_size_{_element_size}, ndims_(shape.size()) {

    std::copy(shape.cbegin(), shape.cend(), std::begin(sizes_));
    std::copy(strides.cbegin(), strides.cend(), std::begin(strides_));
  }

  C10_HOST_DEVICE int64_t index_to_offset(int32_t idx) const {
    int64_t offset = 0;

    #pragma unroll
    for (int32_t dim = 0; dim < 25; ++dim) {
      if (dim == ndims_) {
        break;
      }

      const auto quot = sizes_[dim] / idx;
      const auto rem = sizes_[dim] % idx;

      idx = quot;
      offset += rem * strides_[dim];
    }

    return offset;
  }

  int64_t element_size_;
  int32_t ndims_;
  int32_t sizes_[25];
  int64_t strides_[25];
};

static auto cuda_template = torch::jit::CodeTemplate(R"(
  struct TensorAccessor {
    TensorAccessor() = default;

    // TODO: add a real function here
    __host__ __device__ long index_to_offset(int idx) const {
      return idx;
    }

    long element_size_;
    int ndims_;
    int sizes_[25];
    long strides_[25];
  };

  ${function}

  extern "C" __global__
  void foo_kernel(
      long numel,
      TensorAccessor* out_accessor,
      TensorAccessor* a_accessor,
      TensorAccessor* b_accessor,
      float* out,
      float* a,
      float* b) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
      for (int i = 0; i < 4; ++i) {
        int out_offset = out_accessor->index_to_offset(i);
        int a_offset = a_accessor->index_to_offset(i);
        int b_offset = b_accessor->index_to_offset(i);
        // TODO: allow for custom names
        out[out_offset] = foo(a[a_offset], b[b_offset]);
      }
    }
  }

  #define NUM_THREADS (C10_WARP_SIZE * 2)
  #define THREAD_WORK_SIZE 4
  #define BLOCK_WORK_SIZE (THREAD_WORK_SIZE * num_threads)

  extern "C" __global__
  void vectorized_elementwise_kernel(int numel, ${args}) {
    const int remaining = numel - BLOCK_WORK_SIZE * blockIdx.x;

    if (remaining < BLOCK_WORK_SIZE) {

    } else {
      int idx = blockIdx.x;
      using vec_t = aligned_vector<scalar_t, vec_size>;
      vec_t *from_ = reinterpret_cast<vec_t *>(from);
      int thread_idx = threadIdx.x;
      #pragma unroll
      for (int i = 0; i < loop_size; i++) {
        int index = thread_idx + i * num_threads;
        vec_t v = from_[index];
        #pragma unroll
        for (int j = 0; j < vec_size; j++) {
          to(vec_size * i + j) = v.val[j];
        }
      }
    }
  }


// instantiations here
)");

} // anonymous namespace

Tensor foo_cuda(const Tensor& self, const Tensor& other) {
  Tensor result;
  auto iter = TensorIterator::binary_op(result, self, other);

  std::cout << "dtype 0: " << iter.dtype(0) << std::endl;
  std::cout << "dtype 1: " << iter.dtype(0) << std::endl;
  std::cout << "dtype 2: " << iter.dtype(0) << std::endl;
  std::cout << "iter.tensor(0).scalar_type(): " << iter.tensor(0).scalar_type() << std::endl;
  std::cout << "iter.tensor(1).scalar_type(): " << iter.tensor(1).scalar_type() << std::endl;
  std::cout << "iter.tensor(2).scalar_type(): " << iter.tensor(2).scalar_type() << std::endl;
  std::cout << "common_dtype: " << iter.common_dtype() << std::endl;

  // launch_vectorized_kernel path
  int64_t numel = iter.numel();
  int64_t grid = (numel + block_work_size - 1) / block_work_size;

  const auto ntensors = iter.ntensors();
  // at::detail::Array<char*, ntensors> data;
  // for (auto i = decltype(ntensors){0}; i < ntensors; i++) {
  //   data[i] = (char*)iter.data_ptr(i);
  // }
  // TODO: revise vectorize functions (see MemoryAccess.cuh) to work at runtime
  //   without array allocation
  // int32_t vec_size = memory::can_vectorize_up_to<func_t>(data);
  // TODO: for now assume in case 4
  auto stream = at::cuda::getCurrentCUDAStream();

  // vectorized_elementwise_kernel switch here


  std::vector<void*> args;
  int64_t numel = iter.numel();
  args.push_back((void*)&numel);

  std::cout << "iter.ntensors(): " << iter.ntensors() << std::endl;

  #define stringify(...) std::string("__device__ __forceinline__ " #__VA_ARGS__)
  const auto s = stringify(
    float foo(float a, float b) {
      return a + b;
    }
  );
  #undef stringify

  std::cout << "s: " << s << std::endl;

  torch::jit::TemplateEnv env;
  env.s("function", s);
  std::string code = cuda_template.format(env);

  std::cout << "code: " << code << std::endl;

  std::vector<TensorAccessor> accessors;
  for (auto i = decltype(iter.ntensors()){0}; i < iter.ntensors(); ++i) {
    accessors.emplace_back(iter.shape(), iter.strides(i), iter.element_size(i));
  }

  for (const auto& accessor : accessors) {
    args.push_back((void*)&accessor);
  }

  // Acquires device and NVRTC properties (for compile arch and occupancy
  // calculations)
  hipDeviceProp_t* prop = at::cuda::getCurrentDeviceProperties();
  int major, minor;
  getMajorMinor(prop, major, minor);

  // Creates the NVRTC program
  hiprtcProgram program;
  const auto& nvrtc = at::globalContext().getNVRTC();
  AT_CUDA_NVRTC_CHECK(nvrtc.hiprtcCreateProgram(
      &program, code.c_str(), nullptr, 0, nullptr, nullptr));

  // constructs nvrtc arguments
  const std::string compute = "--gpu-architecture=compute_" +
    std::to_string(major) + std::to_string(minor);
  const std::vector<const char*> build_args = {
    "--std=c++14", compute.c_str(), "-default-device"};

  const auto compilation_result =
        nvrtc.hiprtcCompileProgram(program, build_args.size(), build_args.data());

  if (compilation_result != HIPRTC_SUCCESS) {
    size_t logsize;
    AT_CUDA_NVRTC_CHECK(nvrtc.hiprtcGetProgramLogSize(program, &logsize));
    std::vector<char> log(logsize);
    AT_CUDA_NVRTC_CHECK(nvrtc.hiprtcGetProgramLog(program, log.data()));
    std::stringstream cu;
    cu << log.data();
    throw std::runtime_error(cu.str());
  }

  hipModule_t module;
  hipFunction_t function;
  ::torch::jit::ResourceGuard holdProgram([&] { nvrtc.hiprtcDestroyProgram(&program); });
  std::vector<char> ptx;
  size_t ptx_size;
  AT_CUDA_NVRTC_CHECK(nvrtc.hiprtcGetCodeSize(program, &ptx_size));
  ptx.resize(ptx_size);
  AT_CUDA_NVRTC_CHECK(nvrtc.hiprtcGetCode(program, ptx.data()));

  AT_CUDA_DRIVER_CHECK(nvrtc.hipModuleLoadData(&module, ptx.data()));
  const std::string name = "foo_kernel";
  AT_CUDA_DRIVER_CHECK(
    nvrtc.hipModuleGetFunction(&function, module, name.c_str()));

  int maxBlocks;
  AT_CUDA_DRIVER_CHECK(nvrtc.hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(
    &maxBlocks, function, 128, 0));
  maxBlocks *= prop->multiProcessorCount;

  // const auto nBlocks = std::min(maxBlocks_, ceilDiv(numel, kBlockSize));
  const int nBlocks = 1;

  constexpr int32_t kBlockSize = 128;

  void* out_ptr = iter.output().data_ptr();
  void* self_ptr = self.data_ptr();
  void* other_ptr = other.data_ptr();

  // args.push_back(out_ptr);
  args.push_back((void*)&out_ptr);
  args.push_back((void*)&self_ptr);
  args.push_back((void*)&other_ptr);

  // Launches kernel on current stream (device was set by executor)

  AT_CUDA_DRIVER_CHECK(nvrtc.hipModuleLaunchKernel(
    function,
    nBlocks,
    1,
    1,
    kBlockSize,
    1,
    1,
    0,
    stream,
    args.data(),
    nullptr));

  return iter.output();

  // NOTE: may need/want to initialize CUDA context here (refactor into nvrtc call)

  // void* out, void* a, void* b
  // TODO: provide code (a std::string)
  // const std::string name{"foo_kernel"};
  // const std::string code{R"foo(
  // extern "C" __global__
  // void foo_kernel(void* out, void* a, void* b) {
  //   // if (blockIdx.x == 0 && threadIdx.x == 0) {
  //   //   printf("%f\n", a);
  //   //   printf("%i\n", b);
  //   //   printf("%f\n", ((float*)ptr)[0]);
  //   // }
  //   float* out_float = static_cast<float*>(out);
  //   float* a_float = static_cast<float*>(a);
  //   float* b_float = static_cast<float*>(b);

  //   if (blockIdx.x == 0 && threadIdx.x == 0) {
  //     *out_float = *a_float + *b_float;
  //   }
  // })foo"};
}

}} // namespace at::native
